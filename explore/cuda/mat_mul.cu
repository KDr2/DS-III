
#include <hip/hip_runtime.h>
#include <stdio.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;


void print_matrix(const Matrix mat) {
    for(int r=0; r<mat.height; ++r) {
        for(int c=0; c<mat.width; ++c) {
            printf("%.2f\t", mat.elements[mat.width * r + c]);
        }
        printf("\n");
    }
}

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE

void MatMul(const Matrix A, const Matrix B, Matrix C) {
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); // 16x16
    // result shape, e.g. mat_C.shape is (A.height x b.width)
    dim3 dimGrid((B.width + dimBlock.x -1) / dimBlock.x, (A.height + dimBlock.y -1) / dimBlock.y); // 2x2
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= C.height || col >= C.width) return; // this thread should not do any computing
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}


int main(int argc, char*argv[]) {
    Matrix mat_A;
    mat_A.height = 30;
    mat_A.width = 40;
    mat_A.elements = (float*)malloc(1200 * sizeof(float));
    for(int i=0; i<1200; ++i)mat_A.elements[i] = 1.0;

    Matrix mat_B;
    mat_B.height = 40;
    mat_B.width = 30;
    mat_B.elements = (float*)malloc(1200 * sizeof(float));
    for(int i=0; i<1200; ++i)mat_B.elements[i] = 1.0;

    Matrix mat_C;
    mat_C.width = 30;
    mat_C.height = 30;
    mat_C.elements = (float*)malloc(900 * sizeof(float));

    MatMul(mat_A, mat_B, mat_C);

    print_matrix(mat_C);
    return 0;
}